
#include <hip/hip_runtime.h>
#include <stdio.h>

// __device__ float lerp1d(int a, int b, float w)
// {
//     if(b>a){
//         return a + w*(b-a);
//     }
//     else{
//         return b + w*(a-b);
//     }
// }

__device__ double lerp1d(int a, int b, float w)
{
    return fma(w, (float)b, fma(-w,(float)a,(float)a));
}

__device__ double lerp2d(int f00, int f01, int f10, int f11,
                        float centroid_h, float centroid_w )
{
    centroid_w = (1 + lroundf(centroid_w) - centroid_w)/2;
    centroid_h = (1 + lroundf(centroid_h) - centroid_h)/2;
    
    double r0, r1, r;
    r0 = lerp1d(f00,f01,centroid_w);
    r1 = lerp1d(f10,f11,centroid_w);

    r = lerp1d(r0, r1, centroid_h); //+ 0.00001
    // printf("%f, %f | %f, %f | %f | %d, %d, %d, %d \n", centroid_h , centroid_w, r0, r1, r, f00, f01, f10, f11);
    return r;
}

__global__ void GPU_validation(void)
{
    printf("GPU has been activated \n");
}

__global__ void cuRESIZE(unsigned char* src_img, unsigned char* dst_img, 
    const int src_h, const int src_w, 
    const int dst_h, const int dst_w,
    const float scale_h, const float scale_w)
{
    /* 
    Input: 
        src_img - NHWC
        channel C, default = 3 
    
    Output:
        dst_img - NHWC

    */

    // int const N = gridDim.y; // batch size
    int const n = blockIdx.y; // batch number
    int const C = gridDim.z; // channel 
    int const c = blockIdx.z; // channel number
    long idx = n * blockDim.x * gridDim.x * C + 
               threadIdx.x * gridDim.x * C +
               blockIdx.x * C +
               c;
    
    // some overhead threads in each image process
    // when thread idx in one image exceed one image size return;
    if (idx%(blockDim.x * gridDim.x * C) >= dst_h* dst_w * C){return;} 

    /*
    Now implementation : 
    ( (1024 * int(DST_SIZE/3/1024)+1) - (src_h * src_w) )* N
    = overhead * N times
    
    to do: put the batch into gridDim.x
    dim3 dimGrid(int(DST_SIZE*batch/3/1024)+1,1,3);

    */

    int H = dst_h;
    int W = dst_w;

    int img_coor = idx % (dst_h*dst_w*C); //coordinate of one image, not idx of batch image
    int h = img_coor / (W*C); // dst idx 
    int w = img_coor % (W*C)/C; // dst idx

    float centroid_h, centroid_w;  
    centroid_h = scale_h * (h + 0.5); // h w c -> x, y, z : 1080 , 1920 , 3
    centroid_w = scale_w * (w + 0.5); // 

    // unsigned long = 4,294,967,295 , up to (1080p,RGB)*600 imgs
    long f00,f01,f10,f11;

    int src_h_idx = lroundf(centroid_h)-1;
    int src_w_idx = lroundf(centroid_w)-1;
    if (src_h_idx<0){src_h_idx=0;} // handle boundary pixle
    if (src_w_idx<0){src_w_idx=0;} // handle boundary pixle
    // printf("h:%d w:%d\n",src_h_idx,src_w_idx);
    // printf("src_h_idx:%d , h: %d | src_w_idx:%d , w: %d\n",src_h_idx,h,src_w_idx,w);

    // idx = NHWC = n*(HWC) + h*(WC) + w*C + c;
    f00 = n * src_h * src_w * C + 
          src_h_idx * src_w * C + 
          src_w_idx * C +
          c;
    f01 = n * src_h * src_w * C +
          src_h_idx * src_w * C +
          (src_w_idx+1) * C +
          c;
    f10 = n * src_h * src_w * C +
          (src_h_idx+1) * src_w * C +
          src_w_idx * C +
          c;
    f11 = n * src_h * src_w * C + 
          (src_h_idx+1) * src_w * C +
          (src_w_idx+1) * C +
          c;
    int rs;   
    if (src_w_idx+1>=src_w){f01 = f00; f11 = f10;} // handle boundary pixle
    if (src_h_idx+1>=src_h){f10 = f00; f11 = f01;} // handle boundary pixle

    if (int(f10/ (src_h * src_w * C)) > n ){
        centroid_w = (1 + lroundf(centroid_w) - centroid_w)/2;
        rs = lroundf(lerp1d(f00,f01,centroid_w));
    }else{
        rs = lroundf(lerp2d(src_img[f00], src_img[f01], src_img[f10], src_img[f11], 
            centroid_h, centroid_w));
    }
    
    long dst_idx = n * (H * W * C) + 
                    h * (W * C) +
                    w * C +
                    c;

    dst_img[dst_idx] = (unsigned char)rs;
}

int main(){
    int SRC_HEIGHT = 20;
    int SRC_WIDTH = 20;
    int SRC_SIZE = SRC_HEIGHT * SRC_WIDTH * 3;

    int DST_HEIGHT = 40;
    int DST_WIDTH = 40;
    int DST_SIZE = DST_HEIGHT * DST_WIDTH * 3;

    int batch = 1;
    

    // cudaStream_t stream1, stream2, stream3, stream4 ;
    hipStream_t stream1;
    hipStreamCreate ( &stream1) ;
    
    dim3 dimBlock(1024, 1,1); // maximum threads: 1024
    dim3 dimGrid(int(DST_SIZE/3/1024)+1,batch,3);
    
    unsigned char host_src[SRC_SIZE];
    // unsigned char host_dst[1108992];
    unsigned char host_dst[DST_SIZE];

    // init src image
    for(int i = 0; i < SRC_SIZE; i++){
        host_src[i] = i+1;
        // host_src[i] = (i%3);
    }

    float scale_h = (float)SRC_HEIGHT / DST_HEIGHT;
    float scale_w = (float)SRC_WIDTH / DST_WIDTH;

    unsigned char *device_src, *device_dst;
	hipMalloc((unsigned char **)&device_src, SRC_SIZE* sizeof(unsigned char));
    hipMalloc((unsigned char **)&device_dst, DST_SIZE* sizeof(unsigned char));
    
	hipMemcpy(device_src , host_src , SRC_SIZE * sizeof(unsigned char), hipMemcpyHostToDevice);

    GPU_validation<<<1,1>>>();
    hipDeviceSynchronize();


    cuRESIZE<<<dimGrid, dimBlock, 0, stream1>>>(device_src, device_dst, 
                                                SRC_HEIGHT, SRC_WIDTH,
                                                DST_HEIGHT, DST_WIDTH,
                                                scale_h, scale_w);

    hipDeviceSynchronize();


    // for(int i = 0; i<10; i++){
    // tester<<<dimGrid, dimBlock>>>(device_src, device_dst, 
    //                               SRC_HEIGHT, SRC_WIDTH,
    //                               scale_h, scale_w);
    // cudaDeviceSynchronize();
    // }
    
    hipMemcpy(host_dst, device_dst, DST_SIZE * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // DEBUG : print first image in batch , first 30 pixel in 3 channels.

    // for(int i = 0; i < 30*3; i+=3){ // NHWC
    //     printf("%d\n",host_src[i]);
    // }
    printf("============================\n");
 
    for(int c = 0; c<3*DST_HEIGHT*DST_WIDTH ; c+=DST_HEIGHT*DST_WIDTH){ // if NCHW
        for(int i = 0 ; i < 30; i++){
            printf("%d %d %d\n", c+i, i, host_dst[c+i]);
        }
        printf("------------------------------\n");
    }

    // print first 30 elements from each chanel
    // for(int c = 0; c<3; c++){ // NHWC
    //     for(int i = 0 ; i < 30; i++){
    //         int idx = i*3 +c;
    //         printf("%d %d %d\n", c+i*3, i, host_dst[idx]);
    //     }
    //     printf("------------------------------\n");
    // }

    // int count_0=0;
    // int count_1=0;
    // int count_2=0;
    // for(int idx = 0; idx<sizeof(host_dst)/sizeof(unsigned char); idx++){ // NHWC
    //     printf("%d %d\n", idx, host_dst[idx]);
    //     if (host_dst[idx]==0){count_0++;}
    //     if (host_dst[idx]==1){count_1++;}
    //     if (host_dst[idx]==2){count_2++;}
    // }
    // printf("%d, %d, %d\n",count_0,count_1,count_2);
    // printf("%ld \n",sizeof(host_dst)/sizeof(unsigned char));

	hipFree(device_src);
	hipFree(device_dst);

    return 0;
}
// clear && nvcc resize_free.cu -o resize_free.o && ./resize_free.o